#include "hip/hip_runtime.h"
/*** header ***/


#include <cstdlib>
#include <stdio.h>
#include <math.h>
#include <gd.h>

#include <assert.h>
#include "hip/hip_runtime_api.h"

//#define DO_DITHER
/*** maths ***/

inline double __host__ __device__ dmin(double a,double b) { return a<b ? a:b; }

struct XYZ
{
	double d[3];
	inline void __host__ __device__ Set(double a,double b, double c) { d[0]=a; d[1]=b; d[2]=c; }
	#define do_op(o) \
	inline void __device__ __host__ operator o##= (const XYZ&b) { for(unsigned n=0;n<3;++n) d[n] o##= b.d[n]; } \
	inline void __device__ __host__ operator o##= (double b) { for(unsigned n=0;n<3;++n) d[n] o##= b; } \
	XYZ __device__ __host__ operator o (const XYZ &b) const { XYZ tmp(*this); tmp o##= b; return tmp; }	\
	XYZ __device__ __host__ operator o (double b) const { XYZ tmp(*this); tmp o##= b; return tmp; }
	do_op(*)
	do_op(+)
	do_op(-)
	#undef do_op
	XYZ __device__ __host__ operator- () const { XYZ tmp={{-d[0],-d[1],-d[2]}}; return tmp; }
	XYZ __device__ __host__ Pow(double b) const {XYZ tmp { {pow(d[0],b),pow(d[1],b),pow(d[2],b)}}; return tmp; }
	inline __device__ __host__ double Dot(const XYZ &b) const { return d[0]*b.d[0]+d[1]*b.d[1]+d[2]*b.d[2]; }
	inline __device__ __host__ double Squared() const { return Dot(*this); }
	inline __device__ __host__ double Len() const { return sqrt(Squared()); }
	inline __device__ __host__ void Normalise() { *this *= 1.0/Len(); }
	inline __device__ __host__ void MirrorAround(const XYZ &axis){
		XYZ N=axis;N.Normalise();
		double v=Dot(N);
		*this=N*(v+v)- *this;
	}
	// colour
	inline __device__ __host__ double Luma() const { return d[0]*0.299+d[1]*0.587+d[2]*0.114; }
	inline void __device__ __host__ Clamp()
	{
		for (unsigned n=0;n<3;++n)
		{
			if (d[n]<0.0) d[n]=0.0;
			else if (d[n]>1.0) d[n]=1.0;
		}
	}
	inline void __device__ __host__ ClampWithDesaturation()
	{
		double l=Luma(),sat=1.0;
		if (l>1.0) {d[0]=d[1]=d[2]=1.0;return;}
		if (l<0.0) {d[0]=d[1]=d[2]=0.0;return;}
		for (int n=0;n<3;++n)
			if (d[n]>1.0) sat = dmin(sat,(l-1.0)/(1.0-d[n]));
			else if (d[n]<0.0) sat = dmin(sat,l/(1.0-d[n]));
		if (sat !=1.0)
		{ *this =(*this-1)*sat+l;Clamp(); }
	}
};

struct Matrix
{
	XYZ m[3];
	inline void __device__ __host__ InitRotate(const XYZ &angle)
	{
		double Cx = cos(angle.d[0]), Cy = cos(angle.d[1]), Cz = cos(angle.d[2]);
		double Sx = sin(angle.d[0]), Sy = sin(angle.d[1]), Sz = sin(angle.d[2]);
		double sxsz=Sx*Sz,cxsz=Cx*Sz,cxcz=Cx*Cz,sxcz=Sx*Sz;
		Matrix result = {{ {{Cy*Cz,Cy*Sz,-Sy}},
			{{sxcz*Sy-cxsz,sxsz*Sy+cxcz,Sx*Cy}},
			{{cxcz*Sy+sxsz,cxsz*Sy-sxcz,Cx*Cy}} }};
		*this=result;
	}
	inline void __device__ __host__ Transform(XYZ &vec)
	{
		vec.Set(m[0].Dot(vec),m[1].Dot(vec),m[2].Dot(vec));
	}
};

/*** Walls and Spheres ***/
extern "C" {
// Planes/walls represented by a normal vector and a distance
typedef struct Plane
{XYZ normal;double offset;} Pl;

// Define some for scene

const Pl PlanesPreinit[] = {
	{ {{0,0,-1}}, -30 },
	{ {{0,1,0}}, -30 },
	{ {{0,-1,0}}, -30 },
	{ {{1,0,0}}, -30 },
	{ {{0,0,1}}, -30 },
	{ {{-1,0,0}}, -30 }
};
__device__ __constant__ Pl Planes[sizeof(PlanesPreinit)/sizeof(*PlanesPreinit)];
typedef struct Sphere
{ XYZ center; double radius; } Sp;

// Define some spheres
const Sp SpheresPreinit[] = {
	{ {{0,0,0}}, 7 },
    	{ {{19.4, -19.4, 0}}, 2.1 },
    	{ {{-19.4, 19.4, 0}}, 2.1 },
    	{ {{13.1, 5.1, 0}}, 1.1 },
    	{ {{ -5.1, -13.1, 0}}, 1.1 },
    	{ {{-30,30,15}}, 11},
    	{ {{15,-30,30}}, 6},
    	{ {{30,15,-30}}, 6}
};
__device__ __constant__ Sp Spheres[sizeof(SpheresPreinit)/sizeof(*SpheresPreinit)];

typedef struct LightSource
{XYZ location,colour;} Ls;

const Ls LightsPreinit[] =
{
    { {{-28,-14, 3}}, {{.4, .51, .9}} },
    { {{-29,-29,-29}}, {{.95, .1, .1}} },
    { {{ 14, 29,-14}}, {{.8, .8, .8}} },
    { {{ 29, 29, 29}}, {{1,1,1}} },
    { {{ 28,  0, 29}}, {{.5, .6,  .1}} }
};

__device__ __constant__ Ls Lights[sizeof(LightsPreinit)/sizeof(*LightsPreinit)];

#define NElems(x) sizeof(x)/sizeof(*x)
const unsigned
	NumPlanes = NElems(Planes),
	NumSpheres = NElems(Spheres),
	NumLights = NElems(Lights),
	MAXTRACE = 6;
} // extern "C"
/*** raytracing ***/

int __device__ RayFindObstacle 
	(const XYZ &eye, const XYZ &dir, double &HitDist,
	 int &HitIndex, XYZ & HitLoc, XYZ &HitNormal)
{
	int HitType=-1;
	{for (unsigned i=0;i<NumSpheres;++i)
	{
		XYZ V (eye-Spheres[i].center);
		double r=Spheres[i].radius,
		       DV = dir.Dot(V),
		       D2 = dir.Squared(),
		       SQ = DV*DV - D2*(V.Squared()-r*r);
		// if ray coincides with sphere
		if (SQ<1e-6) continue;
		double SQt = sqrt(SQ),
		       Dist = dmin(-DV-SQt, SQt-DV)/D2;
		if (Dist <1e-6||Dist>=HitDist) continue;
		HitType=1; HitIndex=i;
		HitDist=Dist;
		HitLoc = eye +(dir*HitDist);
		HitNormal = (HitLoc-Spheres[i].center)*(1/r);

	}}
	{for (unsigned i=0; i<NumPlanes;++i)
	{
		double DV =-Planes[i].normal.Dot(dir);
		if (DV>1e-6) continue;
		double D2=Planes[i].normal.Dot(eye),
		       Dist =(D2+Planes[i].offset)/DV;
		if (Dist<1e-6||Dist>=HitDist) continue;
		HitType=0;HitIndex=i;
		HitDist=Dist;
		HitLoc=eye+(dir*HitDist);
		HitNormal=-Planes[i].normal;

	}}
	return HitType;
};


bool __device__ RayFindObstacle(const XYZ& eye, const XYZ& dir, const double HitDist)
{
    // Try intersecting the ray with
    // each object and see which one
    // produces the closest hit.
    int result = 0;
   {for(unsigned i=0; i<NumSpheres; ++i)
    {
        XYZ V (eye - Spheres[i].center);
        double r = Spheres[i].radius,
            DV = dir.Dot(V),
            D2 = dir.Squared(),
            SQ = DV*DV
               - D2*(V.Squared() - r*r);
        // Does the ray coincide
        // with the sphere?
        // Determine where exactly
        double Dist = SQ >= 1e-6 ? dmin(-DV-sqrt(SQ), -DV+sqrt(SQ)) / D2 : 0;
        result |= !(Dist<1e-6 || Dist >= HitDist);
    }}
   {for(unsigned i=0; i<NumPlanes; ++i)
    {
        double DV = -Planes[i].normal.Dot(dir);
        double D2 =
            Planes[i].normal.Dot(eye),
            Dist = (D2+Planes[i].offset) / DV;
        result |= !(Dist<1e-6 || Dist>=HitDist);
    }}
    return result;
}

//extern "C"{
const unsigned NumAreaLightVectors=20;
XYZ AreaLightVectors[NumAreaLightVectors];
//}//extern "C"
void InitAreaLightVectors()
{
	// smooth shadows with cloud of lighsources around point
	for (unsigned i=0;i<NumAreaLightVectors;++i)
		for (unsigned n=0;n<3;++n)
			AreaLightVectors[i].d[n]= 2.0*(rand()/double(RAND_MAX)-0.5)*0.1;
}

//Shoot camera rays
void __device__ RayTrace(XYZ &resultcolour, const XYZ &eye, const XYZ &dir,int k)
{
	double HitDist=1e6;
	XYZ HitLoc,HitNormal;
	int HitIndex,HitType;
	HitType=RayFindObstacle(eye, dir, HitDist, HitIndex, HitLoc, HitNormal);
	// if hits an obs
	if (HitType!=-1)
	{
		XYZ DiffuseLight={{0,0,0}},SpecularLight={{0,0,0}};
		XYZ Pigment {{1,0.98,0.98}};
		for (unsigned i=0;i<NumLights;++i)
			for (unsigned j=0;j<NumAreaLightVectors;++j)
			{
				XYZ V((Lights[i].location+AreaLightVectors[j])-HitLoc);
				double LightDist =V.Len();
				V.Normalise();
				double DiffuseEffect=HitNormal.Dot(V)/(double)NumAreaLightVectors;
				double Attention =(1.0+pow(LightDist/34.0,2.0));
				DiffuseEffect /= Attention;
				if (DiffuseEffect>1e-3)
				{
					double ShadowDist = LightDist-1e-4;
					XYZ a,b;
					int q,t = RayFindObstacle(HitLoc+V*1e-4,V, ShadowDist, q, a, b);
					if (t==-1)
						DiffuseLight += Lights[i].colour*DiffuseEffect;
				}
			}
		if (k>1)
		{
			XYZ V(-dir);V.MirrorAround(HitNormal);
			RayTrace(SpecularLight, HitLoc+V*1e-4, V, k-1);
		}
		switch (HitType) {
			case 0: //Plane
				DiffuseLight *= 0.9;
				SpecularLight *=0.5;
				switch (HitIndex % 3) {
					case 0: Pigment.Set(0.9, 0.7, 0.6); break;
					case 1: Pigment.Set(0.6, 0.7, 0.7); break;
					case 2: Pigment.Set(0.5, 0.8, 0.3); break;
				}
				break;
			case 1: //sphere
				DiffuseLight*=1.0;
				SpecularLight*=0.34;
		}
		resultcolour =(DiffuseLight+SpecularLight)*Pigment;
	}
};

/*** Colour ***/
const double Gamma = 2.0, Ungamma = 1.0 / Gamma;
#ifdef DO_DITHER
extern "C" {
const unsigned CandCount = 64;

unsigned char Dither8x8_init[8][8];
XYZ Pal[16], PalG_init[16];
double lumainit[16];
__device__ __constant__ unsigned char Dither8x8[8][8];
__device__ __constant__ XYZ PalG[16];
__device__ __constant__ double luma[16];
} // extern "C"
void InitDither()
{
    // We will use the default 16-colour EGA/VGA palette.

    for(unsigned i=0; i<16; ++i)
    {
        static const char s[16*3] =
            {0,0,0, 0,0,42, 0,42,0, 0,42,42, 42,0,0, 42,0,42, 42,21,0, 21,21,21,
             42,42,42, 21,21,63, 21,63,21, 21,63,63, 63,21,21, 63,21,63, 63,63,21, 63,63,63};
        Pal[i].Set(s[i*3+0],s[i*3+1],s[i*3+2]);
        Pal[i] *= 1/63.0;
        PalG_init[i] = Pal[i].Pow(Gamma);
        lumainit[i] = PalG_init[i].Luma();
    }
    // Create bayer dithering matrix, adjusted for candidate count
    for(unsigned y=0; y<8; ++y)
        for(unsigned x=0; x<8; ++x)
        {
            unsigned i = x ^ y, j;
            j = (x & 4)/4u + (x & 2)*2u + (x & 1)*16u;
            i = (i & 4)/2u + (i & 2)*4u + (i & 1)*32u;
            Dither8x8_init[y][x] = (j+i)*CandCount/64u;
        }
}
#endif

/*** main ***/
const unsigned W = 1920, H = 1080;
const unsigned Threads = 256;
const unsigned Blocks  = (W*H + (Threads-1)) / Threads;

void __global__ RenderScreen(
		#ifdef DO_DITHER
		unsigned char* results,
		#else
		unsigned* results
		#endif
		double camanglex,double camangley,double camanglez,
                double camlookx,double camlooky,double camlookz,
                double zoom,
                double contrast,double contrast_offset)
{
	unsigned pixno = blockIdx.x * blockDim.x + threadIdx.x;
	if(pixno >= W*H) return;
	XYZ camangle = { { camanglex,camangley,camanglez } };
    	XYZ camlook = { { camlookx,camlooky,camlookz } };
    	XYZ campos = { { 0.0, 0.0, 16.0} };
	Matrix camrotatematrix, camlookmatrix;
    	camrotatematrix.InitRotate(camangle);
    	camrotatematrix.Transform(campos);
    	camlookmatrix.InitRotate(camlook);
	const unsigned x = pixno % W;
    	const unsigned y = pixno / W;
    	XYZ camray = { { x / double(W) - 0.5,
                     	y / double(H) - 0.5,
                     	zoom } };
    	camray.d[0] *= double(W)/double(H); // Aspect ratio correction
	camray.Normalize();
    	camlookmatrix.Transform(camray);
    	XYZ campix;
    	RayTrace(campix, campos, camray, MAXTRACE);
    	campix *= 0.5;
    	resluma[y*W+x] = campix.Luma();
    	// Exaggerate the colors to bring contrast better forth
    	campix = (campix + contrast_offset) * contrast;
    	// Clamp, and compensate for display gamma (for dithering)
    	campix.ClampWithDesaturation();
    	XYZ campixG = campix.Pow(Gamma);
	#ifdef DO_DITHER
    	XYZ qtryG = campixG;
    	// Create candidate for dithering
    	unsigned candlist[CandCount];
    	for(unsigned i=0; i<CandCount; ++i)
    	{
        	unsigned k = 0;
        	double b = 1e6;
        	// Find closest match from palette
        	for(unsigned j=0; j<16; ++j)
        	{
            		double a = (qtryG - PalG[j]).Squared();
            		if(a < b) { b = a; k = j; }
        	}
        	candlist[i] = k;
        	if(i+1 >= CandCount) break;
        	// Compensate for error
        	qtryG += (campixG - PalG[k]);
        	qtryG.Clamp();
    	}
    	// Order candidates by luminosity
    	// using insertion sort.
    	for(unsigned j=1; j<CandCount; ++j)
    	{
        	unsigned k = candlist[j], i;
        	for(i=j; i>=1 && luma[candlist[i-1]] > luma[k]; --i)
	            candlist[i] = candlist[i-1];
        candlist[i] = k;
    	}
    	// Draw pixel (use BIOS).
    	results[y*W+x] = candlist[Dither8x8[x & 7][y & 7]];
	#else
    	results[y*W+x] = (unsigned(campixG.d[0] * 255) << 16)
                   + (unsigned(campixG.d[1] * 255) << 8)
                   + (unsigned(campixG.d[2] * 255) << 0);
	#endif
	}
}



int main()
{

    InitAreaLightVectors();
    #define PreInit(symbol, from) checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(symbol), &from, sizeof(from)))
    PreInit(ArealightVectors, ArealightVectorsPreinit);
    #ifdef DO_DITHER
    InitDither();
    PreInit(PalG, PalG_init);
    PreInit(luma, lumainit);
    PreInit(Dither8x8, Dither8x8_init);
    #endif
    PreInit(Planes, PlanesPreinit);
    PreInit(Spheres, SpheresPreinit);
    PreInit(Lights, LightsPreinit);
    #undef PreInit
    checkCudaErrors(hipDeviceSetLimit(hipLimitStackSize,2500));
    XYZ camangle      = { {0,0,0} };
    XYZ camangledelta = { {-.005, -.011, -.017} };
    XYZ camlook       = { {0,0,0} };
    XYZ camlookdelta  = { {-.001, .005, .004} };

    double zoom = 46.0, zoomdelta = 0.99;
    double contrast = 32, contrast_offset = -0.17;

    //const unsigned W = 680, H = 480;

#ifdef DO_DITHER
    static unsigned char results[W*H], *p = NULL;
#else
    static unsigned      results[W*H], *p = NULL;
#endif
    
    static double        resluma[W*H], *L = NULL;
    checkCudaErrors(hipMalloc((void**)&p, sizeof(results))); assert(p!=NULL);
    checkCudaErrors(hipMalloc((void**)&L, sizeof(resluma))); assert(L!=NULL);
    for(unsigned frameno=0; frameno<2048; ++frameno)
    {
        //fprintf(stderr, "Begins frame %u; contrast %g, contrast offset %g\n",
            //frameno,contrast,contrast_offset);
	#ifdef DO_DITHER
        gdImagePtr im = gdImageCreate(W,H);
	
        for(unsigned p=0; p<16; ++p)
            gdImageColorAllocate(im, (int)(Pal[p].d[0]*255+0.5),
                                     (int)(Pal[p].d[1]*255+0.5),
                                     (int)(Pal[p].d[2]*255+0.5));
				     
	#else
        	gdImagePtr im = gdImageCreateTrueColor(W,H);
    	#endif
        // Put camera between the central sphere and the walls
        //XYZ campos = { { 0.0, 0.0, 16.0} };
        // Rotate it around the center
        //Matrix camrotatematrix, camlookmatrix;
        //camrotatematrix.InitRotate(camangle);
        //camrotatematrix.Transform(campos);
        //camlookmatrix.InitRotate(camlook);

        // Determine the contrast ratio for this frame's pixels
        fprintf(stderr, "Begins frame %u; contrast %g, contrast offset %g ",
            frameno,contrast,contrast_offset); fflush(stderr);
	
        RenderScreen<<<Blocks,Threads,0>>> (p,L,
                                            camangle.d[0],camangle.d[1],camangle.d[2],
                                            camlook.d[0],camlook.d[1],camlook.d[2],
                                            zoom,
                                            contrast,contrast_offset);
        checkCudaErrors(hipMemcpy(results, p, sizeof(results), hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(resluma, L, sizeof(resluma), hipMemcpyDeviceToHost));
	double thisframe_min = 100;
        double thisframe_max = -100;

      #pragma omp parallel for collapse(2)
	for(unsigned y=0; y<H; ++y)
            for(unsigned x=0; x<W; ++x)
            {
        	#pragma omp critical
              {
                // Update frame luminosity info for automatic contrast adjuster
                double lum = resluma[y*w+x];
                #pragma omp flush(thisframe_min,thisframe_max)
                if(lum < thisframe_min) thisframe_min = lum;
                if(lum > thisframe_max) thisframe_max = lum;
                #pragma omp flush(thisframe_min,thisframe_max)
              }
                // Exaggerate the colours to bring contrast better forth
             unsigned color = results[y*W+x];
		//int colour = gdTrueColor((int) campix.d[0]*255,(int) campix.d[1]*255, (int) campix.d[2]*255);
                gdImageSetPixel(im, x,y, colour);
            }

        char Buf[64]; sprintf(Buf, "trace%d.png", frameno);
        fprintf(stderr, "Writing %s...\n", Buf);
        FILE* fp = fopen(Buf, "wb");
        gdImagePng(im, fp);
        gdImageDestroy(im);
        fclose(fp);


        // Tweak coordinates / camera parameters for the next frame
        double much = 1.0;

        // In the beginning, do some camera action (play with zoom)
        if(zoom <= 1.1)
            zoom = 1.1;
        else
        {
            if(zoom > 40) { if(zoomdelta > 0.95) zoomdelta -= 0.001; }
            else if(zoom < 3) { if(zoomdelta < 0.99) zoomdelta += 0.001; }
            zoom *= zoomdelta;
            much = 1.1 / pow(zoom/1.1, 3);
        }

        // Update the rotation angle
        camlook  += camlookdelta * much;
        camangle += camangledelta * much;

        // Dynamically readjust the contrast based on the contents
        // of the last frame
        double middle = (thisframe_min + thisframe_max) * 0.5;
        double span   = (thisframe_max - thisframe_min);
        thisframe_min = middle - span*0.60; // Avoid dark tones
        thisframe_max = middle + span*0.37; // Emphasize bright tones
        double new_contrast_offset = -thisframe_min;
        double new_contrast        = 1 / (thisframe_max - thisframe_min);
        // Avoid too abrupt changes, though
        double l = 0.85;
        if(frameno == 0) l = 0.7;
        contrast_offset = (contrast_offset*l + new_contrast_offset*(1.0-l));
        contrast        = (contrast*l + new_contrast*(1.0-l));
    }
    checkCudaErrors(hipFree(p));
    checkCudaErrors(hipFree(L));
}
